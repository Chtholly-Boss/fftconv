#include "fftconv2d.h"

at::Tensor rfft(at::Tensor signal) {
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int rSize = signal.size(0);
	// create empty output tensor
	int cSize = rSize / 2 + 1;
	auto dout_real = torch::empty({cSize}, signal.options());
	auto dout_imag = torch::empty({cSize}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);
	
	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft(at::Tensor signal) {
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int cSize = signal.size(0);

	// create output tensor
	int rSize = (cSize - 1) * 2;
	auto dout = torch::empty({rSize}, signal.options().dtype(torch::kFloat32));

	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2R, 1);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= n;
	return dout;
}

at::Tensor rfft2(at::Tensor signal) {
	// TODO: implement 2D FFT
	// Check input type is kFloat32
	CHECK_INPUT(signal);
	CHECK_REAL(signal);
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");
	// Get Height and Width
	int rHeight = signal.size(0);
	int rWidth = signal.size(1);
	// create empty output tensor
	int cHeight = rHeight;
	int cWidth = rWidth / 2 + 1;

	auto dout_real = torch::empty({cHeight, cWidth}, signal.options());
	auto dout_imag = torch::empty({cHeight, cWidth}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_R2C);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft2(at::Tensor signal) {
    // TODO: implement Inverse 2D FFT
	// Check input type is complex
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");

	// Get Height and Width
	int cHeight = signal.size(0);
	int cWidth = signal.size(1);

	// create output tensor
	int rHeight = cHeight;
	int rWidth = (cWidth - 1) * 2;

	auto dout = torch::empty({rHeight, rWidth}, signal.options().dtype(torch::kFloat32));

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;

	hipfftPlan2d(&plan, x, y, HIPFFT_C2R);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= (rHeight * rWidth);
	return dout;
}

// Declare rfftn takes a Tensor and a vector of ints that specify the dimension to do fft
at::Tensor rfftn(at::Tensor signal) {
	// Check input type is kFloat32
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	// do fft on the last 2 dimensions, treat other dimensino as batch dimension
	int ndim = signal.dim();
	int width = signal.size(ndim - 1);
	int height = signal.size(ndim - 2);

	int cHeight = height;
	int cWidth = width / 2 + 1;
	// Get signal.size() and replace last 2 dimension with cHeight and cWidth
	auto size_tuple = signal.sizes();
	std::vector<int64_t> sizes(size_tuple.begin(), size_tuple.end());

	sizes[ndim - 1] = cWidth;
	sizes[ndim - 2] = cHeight;
	auto dout_real = torch::empty(sizes, signal.options());
	auto dout_imag = torch::empty(sizes, signal.options());
	auto dout = at::complex(dout_real, dout_imag);

	// create 2D cuFFT plan
	int x = height, y = width;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_R2C);
	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// treat other dimensions as batch dimension, compute the batch size
	int batch = 1;
	for (int i = 0; i < ndim - 2; i++) {
		batch *= signal.size(i);
	}
	// perform 2D fft on the last 2 dimension
	for (int i = 0; i < batch; i++) {
		hipfftExecR2C(
			plan, 
			din_fft + i * height * width, 
			dout_fft + i * cHeight * cWidth
		);
	}
	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfftn(at::Tensor signal) {
	// Check input type is complex
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	// Get the parameters
	int ndim = signal.dim();
	int cWidth = signal.size(ndim-1);
	int cHeight = signal.size(ndim-2);

	int rHeight = cHeight;
	int rWidth = (cWidth - 1) * 2;

	auto size_tuple = signal.sizes();
	std::vector<int64_t> sizes(size_tuple.begin(), size_tuple.end());

	sizes[ndim - 1] = rWidth;
	sizes[ndim - 2] = rHeight;
	auto dout = torch::empty(sizes, signal.options().dtype(torch::kFloat32));

	// compute the batch size
	int batch = 1;
	for (int i = 0; i < ndim - 2; i++) {
		batch *= signal.size(i);
	}
	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2R);

	// cast signal to cuFFT type
	// dup the signal to avoid modifying the original signal
	at::Tensor signal_dup = signal.clone();
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal_dup.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// perform 2D ifft on the last 2 dimension
	for (int i = 0; i < batch; i++) {
		hipfftExecC2R(
			plan,
			din_fft + i * cHeight * cWidth,
			dout_fft + i * rHeight * rWidth
		);
	}
	// destroy cuFFT plan
	hipfftDestroy(plan);
	// normalize
	dout /= (rHeight * rWidth);
	return dout;
}