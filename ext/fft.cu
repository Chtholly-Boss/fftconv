#include "cusfft.h"
#include <hipfft/hipfft.h>

at::Tensor rfft(at::Tensor signal) {
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int rSize = signal.size(0);
	// create empty output tensor
	int cSize = rSize / 2 + 1;
	auto dout_real = torch::empty({cSize}, signal.options());
	auto dout_imag = torch::empty({cSize}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);
	
	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft(at::Tensor signal) {
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int cSize = signal.size(0);

	// create output tensor
	int rSize = (cSize - 1) * 2;
	auto dout = torch::empty({rSize}, signal.options().dtype(torch::kFloat32));

	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2R, 1);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= n;
	return dout;
}

at::Tensor rfft2(at::Tensor signal) {
	// TODO: implement 2D FFT
	// Check input type is kFloat32
	CHECK_INPUT(signal);
	CHECK_REAL(signal);
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");
	// Get Height and Width
	int rHeight = signal.size(0);
	int rWidth = signal.size(1);
	// create empty output tensor
	int cHeight = rHeight;
	int cWidth = rWidth / 2 + 1;

	auto dout_real = torch::empty({cHeight, cWidth}, signal.options());
	auto dout_imag = torch::empty({cHeight, cWidth}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_R2C);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft2(at::Tensor signal) {
    // TODO: implement Inverse 2D FFT
	// Check input type is complex
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");

	// Get Height and Width
	int cHeight = signal.size(0);
	int cWidth = signal.size(1);

	// create output tensor
	int rHeight = cHeight;
	int rWidth = (cWidth - 1) * 2;

	auto dout = torch::empty({rHeight, rWidth}, signal.options().dtype(torch::kFloat32));

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;

	hipfftPlan2d(&plan, x, y, HIPFFT_C2R);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= (rHeight * rWidth);
	return dout;
}

// Declare rfftn takes a Tensor and a vector of ints that specify the dimension to do fft
at::Tensor rfftn(at::Tensor signal) {
	// Check input type is kFloat32
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	// Get the parameters
	int batch = signal.size(0);
	int channel = signal.size(1);
	int height = signal.size(2);
	int width = signal.size(3);

	// create empty output tensor
	int cHeight = height;
	int cWidth = width / 2 + 1;

	auto dout_real = torch::empty({batch, channel, cHeight, cWidth}, signal.options());
	auto dout_imag = torch::empty({batch, channel, cHeight, cWidth}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);

	// create 2D cuFFT plan
	int x = height, y = width;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_R2C);
	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());
	// execute cuFFT plan
	// loop on batch and channel
	for (int i = 0; i < batch; i++) {
		for (int j = 0; j < channel; j++) {
			hipfftExecR2C(
				plan, 
				din_fft + i * channel * height * width + j * height * width, 
				dout_fft + i * channel * cHeight * cWidth + j * cHeight * cWidth
			);
		}
	}
	
	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfftn(at::Tensor signal) {
	// Check input type is complex
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	// Get the parameters
	int batch = signal.size(0);
	int channel = signal.size(1);
	int cHeight = signal.size(2);
	int cWidth = signal.size(3);

	// create empty output tensor
	int rHeight = cHeight;
	int rWidth = (cWidth - 1) * 2;
	auto dout = torch::empty({batch, channel, rHeight, rWidth}, signal.options().dtype(torch::kFloat32));

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2R);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());
	// execute cuFFT plan
	// loop on batch and channel
	for (int i = 0; i < batch; i++) {
		for (int j = 0; j < channel; j++) {
			hipfftExecC2R(
				plan, 
				din_fft + i * channel * cHeight * cWidth + j * cHeight * cWidth, 
				dout_fft + i * channel * rHeight * rWidth + j * rHeight * rWidth
			);
			// normalize output tensor
			dout[i][j] /= (rHeight * rWidth);
		}
	}
	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}