#include <torch/extension.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <iostream>

at::Tensor rfft(at::Tensor signal) {
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int rSize = signal.size(0);
	// create empty output tensor
	int cSize = rSize / 2 + 1;
	auto dout_real = torch::empty({cSize}, signal.options());
	auto dout_imag = torch::empty({cSize}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);
	
	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft(at::Tensor signal) {
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int cSize = signal.size(0);

	// create output tensor
	int rSize = (cSize - 1) * 2;
	auto dout = torch::empty({rSize}, signal.options().dtype(torch::kFloat32));

	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2R, 1);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= n;
	return dout;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("rfft", &rfft, "1D FFT(CUDA)");
	m.def("irfft", &irfft, "Inverse 1D FFT(CUDA)");
}
