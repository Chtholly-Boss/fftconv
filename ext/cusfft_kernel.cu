#include <torch/extension.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <iostream>

at::Tensor rfft(at::Tensor signal) {
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int rSize = signal.size(0);
	// create empty output tensor
	int cSize = rSize / 2 + 1;
	auto dout_real = torch::empty({cSize}, signal.options());
	auto dout_imag = torch::empty({cSize}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);
	
	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor irfft(at::Tensor signal) {
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");

	int cSize = signal.size(0);

	// create output tensor
	int rSize = (cSize - 1) * 2;
	auto dout = torch::empty({rSize}, signal.options().dtype(torch::kFloat32));

	// create 1D cuFFT plan
	int n = rSize;
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2R, 1);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= n;
	return dout;
}

at::Tensor rfftn(at::Tensor signal) {
	// TODO: implement 2D FFT
	// Check input type is kFloat32
	TORCH_CHECK(signal.dtype() == torch::kFloat32, "signal must be torch::kFloat32");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");
	// Get Height and Width
	int rHeight = signal.size(0);
	int rWidth = signal.size(1);
	// create empty output tensor
	int cHeight = rHeight;
	int cWidth = rWidth / 2 + 1;

	auto dout_real = torch::empty({cHeight, cWidth}, signal.options());
	auto dout_imag = torch::empty({cHeight, cWidth}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_R2C);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	return dout;
}

at::Tensor irfftn(at::Tensor signal) {
    // TODO: implement Inverse 2D FFT
	// Check input type is complex
	TORCH_CHECK(signal.is_complex(), "signal must be at::complex");
	TORCH_CHECK(signal.is_cuda(), "signal must be on cuda");
	TORCH_CHECK(signal.dim() == 2, "signal must be 2D");

	// Get Height and Width
	int cHeight = signal.size(0);
	int cWidth = signal.size(1);

	// create output tensor
	int rHeight = cHeight;
	int rWidth = (cWidth - 1) * 2;

	auto dout = torch::empty({rHeight, rWidth}, signal.options().dtype(torch::kFloat32));

	// create 2D cuFFT plan
	int x = rHeight, y = rWidth;
	hipfftHandle plan;

	hipfftPlan2d(&plan, x, y, HIPFFT_C2R);

	// cast signal to cuFFT type
	hipfftComplex *din_fft = reinterpret_cast<hipfftComplex *>(signal.data_ptr());
	hipfftReal *dout_fft = reinterpret_cast<hipfftReal *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecC2R(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);

	// normalize output tensor
	dout /= (rHeight * rWidth);
	return dout;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("rfft", &rfft, "1D FFT(CUDA)");
	m.def("irfft", &irfft, "Inverse 1D FFT(CUDA)");
	m.def("rfftn", &rfftn, "N-D FFT(CUDA)");
	m.def("irfftn", &irfftn, "Inverse N-D FFT(CUDA)");
}
