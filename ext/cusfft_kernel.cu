#include <torch/extension.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <iostream>

at::Tensor fft(at::Tensor signal) {
	// check if signal is torch::kFloat32
	if (signal.type().scalarType() != torch::kFloat32) {
		throw std::runtime_error("signal must be torch::kFloat32");
	}
	// check if signal is on cuda
	if (!signal.is_cuda()) {
		throw std::runtime_error("signal must be on cuda");
	}

	int n = signal.size(0);
	// create empty output tensor
	int out_size = n / 2 + 1;
	auto dout_real = torch::empty({out_size}, signal.options());
	auto dout_imag = torch::empty({out_size}, signal.options());
	auto dout = at::complex(dout_real, dout_imag);
	
	// create 1D cuFFT plan
	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);

	// cast signal to cuFFT type
	hipfftReal *din_fft = reinterpret_cast<hipfftReal *>(signal.data_ptr());
	hipfftComplex *dout_fft = reinterpret_cast<hipfftComplex *>(dout.data_ptr());

	// execute cuFFT plan
	hipfftExecR2C(plan, din_fft, dout_fft);

	// destroy cuFFT plan
	hipfftDestroy(plan);
	return dout;
}

at::Tensor rfftn(at::Tensor signal, std::vector<int64_t> dims) {
	// TODO: You will only perform FFT on the specified dim
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("fft", &fft, "1D FFT(CUDA)");
}
